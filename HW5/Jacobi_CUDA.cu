/* Modified from Jessie's reference code */

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define PI 3.14159265359f
#define MAX(a,b) (((a)>(b))?(a):(b))
#define p_Nthreads 32

__global__ void jacobi(int N, float * u, float *f, float *unew){
  
  const int i = threadIdx.x + blockIdx.x*blockDim.x + 1; // offset by 1
  const int j = threadIdx.y + blockIdx.y*blockDim.y + 1;

  if (i < N+1 && j < N+1){
    const int Np = (N+2);
    const int id = i + j*(N+2);
    const float ru = -u[id-Np]-u[id+Np]-u[id-1]-u[id+1];
    const float newu = .25 * (f[id] - ru);
    unew[id] = newu;
  }
}

// use all threads
__global__ void reduce(int N2, float *u, float *unew, float *res){

  __shared__ volatile float s_x[p_Nthreads]; // volatile for in-warp smem mods

  const int tid = threadIdx.x;
  const int i = tid + blockIdx.x*(2*blockDim.x);

  s_x[tid] = 0;
  if (i < N2){
    const float unew1 = unew[i];
    const float unew2 = unew[i + blockDim.x];
    const float diff1 = unew1 - u[i];
    const float diff2 = unew2 - u[i + blockDim.x];
    s_x[tid] = diff1*diff1 + diff2*diff2; 

    // update u
    u[i] = unew1;
    u[i + blockDim.x] = unew2;
  }
  __syncthreads();
  
  // stop at s = 64
  for (unsigned int s = blockDim.x/2; s > 32; s /= 2){
    if (tid < s){
      s_x[tid] += s_x[tid+s]; 
    }
    __syncthreads();
  }   

  // manually reduce within a warp
  if (tid < 32){
    s_x[tid] += s_x[tid + 32];
    s_x[tid] += s_x[tid + 16];
    s_x[tid] += s_x[tid + 8];
    s_x[tid] += s_x[tid + 4];
    s_x[tid] += s_x[tid + 2];
    s_x[tid] += s_x[tid + 1];   
  }
  if (tid==0){
    res[blockIdx.x] = s_x[0];
  }
}


int main(int argc, char **argv){
   
  int N = atoi(argv[1]);
  float tol = atof(argv[2]);

  float *u = (float*) calloc((N+2)*(N+2), sizeof(float));
  float *unew = (float*)calloc((N+2)*(N+2),sizeof(float));
  float *f = (float*) calloc((N+2)*(N+2), sizeof(float));
  float h = 2.0/(N+1);
  for (int i = 0; i < N+2; ++i){
    for (int j = 0; j < N+2; ++j){
      const float x = -1.0 + i*h;
      const float y = -1.0 + j*h;
      f[i + j*(N+2)] = sin(PI*x)*sin(PI*y) * h*h;
    }
  } 

  // cuda memory for Jacobi variables
  float *c_u, *c_f, *c_unew;
  hipMalloc(&c_u, (N+2)*(N+2)*sizeof(float));
  hipMalloc(&c_f, (N+2)*(N+2)*sizeof(float));
  hipMalloc(&c_unew, (N+2)*(N+2)*sizeof(float));
  hipMemcpy(c_u,u, (N+2)*(N+2)*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(c_f,f, (N+2)*(N+2)*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(c_unew,unew,(N+2)*(N+2)*sizeof(float),hipMemcpyHostToDevice);

  // run kernel, copy result back to CPU
  int Nthreads = p_Nthreads; // good if it's a multiple of 32, can't have more than 1024
  int Nblocks = (N + Nthreads-1)/Nthreads; 
  dim3 threadsPerBlock(Nthreads,Nthreads,1);  
  dim3 blocks(Nblocks,Nblocks,1);

  // for reduce kernel
  int Nthreads1D = p_Nthreads; 
  int Nblocks1D = ((N+2)*(N+2) + Nthreads-1)/Nthreads; 
  int halfNblocks1D = (Nblocks1D + 1)/2; 
  dim3 threadsPerBlock1D(Nthreads1D,1,1);  
  dim3 halfblocks1D(halfNblocks1D,1,1);

  // storage for residual
  float *res = (float*) calloc(halfNblocks1D, sizeof(float));
  float *c_res;
  hipMalloc(&c_res, halfNblocks1D*sizeof(float));

  float time, time1=0.f, time2=0.f;
  hipEvent_t start, stop;
  int iter = 0;
  float r2 = 1.;
  while (r2 > tol*tol){

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
    jacobi <<< blocks, threadsPerBlock >>> (N, c_u, c_f, c_unew);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time1 += time;

	hipEventRecord(start,0);
    reduce <<< halfblocks1D, threadsPerBlock1D >>> ((N+2)*(N+2), c_u, c_unew, c_res);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time2 += time;

    // finish block reduction on CPU
    hipMemcpy(res,c_res,halfNblocks1D*sizeof(float),hipMemcpyDeviceToHost);
    r2 = 0.f;
    for (int j = 0; j < halfNblocks1D; ++j){
      r2 += res[j];
    }

	if((!iter%1000)){
		printf("Iter: %d. error = %lg\n", iter, sqrt(r2));
	}
    ++iter;
  }
 
  hipMemcpy(u,c_unew,(N+2)*(N+2)*sizeof(float),hipMemcpyDeviceToHost);

  float err = 0.0;
  for (int i = 0; i < (N+2)*(N+2); ++i){
    err = MAX(err,fabs(u[i] - f[i]/(h*h*2.0*PI*PI)));
  }
  
  printf("Max error: %f, r2 = %lg, iterations = %d\n", err,sqrt(r2),iter);
  printf("Jacobi kernel computing time is: %f milliseconds\n", time1);
  printf("Reduction kernel computing time is: %f milliseconds\n", time2);

}
