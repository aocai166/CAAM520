/*************************************************************************
	> File Name: cuda_Jacobi_shared.cu
	> Author: Ao Cai
	> Mail: aocai166@gmail.com 
	> Created Time: April 04 2019 09:48:38 AM CST
 ************************************************************************/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>

#define p_N 100
#define p_Nthreads 256
#define dfloat float  // switch between double/single precision
#define MAX(a,b) ((a)>(b)?(a):(b))

__global__ void compute_xk(int N, dfloat *u, dfloat *b, dfloat *res)
/* Computing the new xk and send it to u */
{
	__shared__ dfloat s_x[3*p_Nthreads];
	__shared__ dfloat s_b[p_Nthreads];

	const int ii = blockIdx.x*blockDim.x + threadIdx.x;
	const int ix = blockIdx.x;
	const int iz = threadIdx.x;

	dfloat newu = 0.0;

	if(ix>0 && ix<N){
		s_x[iz] = u[ii-N];
		s_x[iz+N] = u[ii];
		s_x[iz+2*N] = u[ii+N];
		s_b[iz] = b[ii];
	}

	__syncthreads();

	if(ix > 0 && ix < N){
		if(iz > 0 && iz < N){

			dfloat invD=0.25, Ru, tmp;
			Ru = 0.0;
			Ru -= (iz-1>=0)?s_x[iz]:0.0;
			Ru -= (iz+1<N)?s_x[iz+2*N]:0.0;
			Ru -= (ix-1>=0)?s_x[iz+N-1]:0.0;
			Ru -= (ix+1<N)?s_x[iz+N+1]:0.0;

			tmp = s_b[iz] - Ru;
			newu = invD*tmp;
			tmp = tmp - 4.0*u[ii];
			res[ii] = tmp*tmp;
		}
	}

	__syncthreads();

	if(ix > 0 && ix < N){
		if(iz > 0 && iz < N){
			u[ii] = newu;
		}
	}
}

__global__ void reduce1(int N, float *x, float *xout){

  __shared__ float s_x[p_Nthreads];

  const int tid = threadIdx.x;
  const int i = blockIdx.x*blockDim.x + tid;

  // load smem
  s_x[tid] = 0;
  if (i < N){
    s_x[tid] = x[i];
  }
  __syncthreads();

  for (unsigned int s = 1; s < blockDim.x; s *= 2){
    int index = 2*s*tid;
    if (index < blockDim.x){
      s_x[index] += s_x[index+s]; // bank conflicts
    }
    __syncthreads();
  }   

  if (tid==0){
    xout[blockIdx.x] = s_x[0];
  }
}

int main(void){
	//int N = atoi(argv[1]);
	int N = p_N;

	int ii, ix, iz;
	dfloat h, tmp, tmpx, tmpz, obj=1.0, tol; // objective function & model difference objective
	dfloat *u, *b, *res; // A is the differential matrix and b is the source function

	printf("N=%d, thread-block size: %d\n",N, p_Nthreads);
	u = (dfloat*) calloc(N*N, sizeof(dfloat));
	b = (dfloat*) calloc(N*N, sizeof(dfloat));
	res = (dfloat*) calloc(N*N, sizeof(dfloat));

//	for(ii=0; ii<N*N; ii++){
//		u[ii] = 1.0;
//	}

	h = 2.0/(N+1.0);
	tmp = h*h;
	tol = 1e-6;

	for(iz = 0; iz < N; iz++){
		for (ix = 0; ix < N; ix++){
			ii = ix + iz*N;

			tmpx = (ix+1.0)*h-1.0;
			tmpz = (iz+1.0)*h-1.0;
			b[ii] = tmp*sin(M_PI*tmpx)*sin(M_PI*tmpz);
		}
	}

	// Allocate CUDA memory
	dfloat *c_u, *c_b, *c_res, *c_out;
	hipMalloc(&c_u, N*N*sizeof(dfloat));
	hipMalloc(&c_b, N*N*sizeof(dfloat));
	hipMalloc(&c_res, N*N*sizeof(dfloat));

	// Copy host memory over to GPU
	hipMemcpy(c_u, u, N*N*sizeof(dfloat), hipMemcpyHostToDevice);
	hipMemcpy(c_b, b, N*N*sizeof(dfloat), hipMemcpyHostToDevice);
	hipMemcpy(c_res, res, N*N*sizeof(dfloat), hipMemcpyHostToDevice);

	// Initialization
	int Nthreads = N;
	int Nblocks = N;
	dim3 threadsPerBlock(Nthreads,1,1);
	dim3 blocks(Nblocks,1,1);

	int Nthreads_reduce = p_Nthreads;
	int Nblocks_reduce = (N*N+Nthreads_reduce-1)/Nthreads_reduce;
	dim3 threadsPerBlock_reduce(Nthreads_reduce,1,1);
	dim3 blocks_reduce(Nblocks_reduce,1,1);

	dfloat *out = (dfloat*)malloc(Nblocks_reduce*sizeof(dfloat));
	hipMalloc(&c_out, Nblocks_reduce*sizeof(dfloat));

	int iter = 0;

	while(obj > tol*tol){
		obj = 0.0;

		compute_xk<<< blocks, threadsPerBlock >>> (N, c_u, c_b, c_res);

		reduce1<<< blocks_reduce, threadsPerBlock_reduce >>> (N*N, c_res, c_out);

		hipMemcpy(out, c_out, Nblocks_reduce*sizeof(dfloat), hipMemcpyDeviceToHost);

		for (ii=0; ii< Nblocks_reduce; ii++){
			obj += out[ii];
		}
		if(!(iter%1000)){
			printf("Iter: %d, error = %lg\n", iter, sqrt(obj));
		}

		iter++;
	}
	printf("%s\n",hipGetErrorString(hipGetLastError()));

	if(N==2)printf("The numerical solution u1=%f u2=%f u3=%f u4=%f\n",u[0],u[1],u[2],u[3]);

	// check result
	dfloat err=0.0;
	for(int ii=0; ii<N*N; ii++){
		err = MAX(err,fabs(u[ii]-b[ii]/(h*h*2.0*M_PI*M_PI)));
	}
	printf("Final Iteration: %d, obj= %lg\n", iter, sqrt(obj));
	printf("Max error: %lg\n", err);

	// free memory on both CPU and GPU
	hipFree(c_u);
	hipFree(c_b);
	hipFree(c_res);
	hipFree(c_out);
	free(u);
	free(b);
	free(res);
	free(out);
}
